#include "hip/hip_runtime.h"
__global__ void conv2d_basic_kernel(float *N, float *F, float *P, int r, int width, int height) {
    int outCol = blockIdx.x * blockDim.x + threadIdx.x;
    int outRow = blockIdx.y * blockDim.y + threadIdx.y;

    float Pvalue = 0;
    int inRow, inCol;

    for (int fRow = 0; fRow < 2*r+1; fRow++) {
        for (int fCol = 0; fCol < 2*r+1; fCol++) {
            inRow = outRow - r + fRow;
            inCol = outCol - r + fCol;

            if (inRow>=0 && inRow < height && inCol >= 0 && inCol < width) {
                Pvalue += F[fRow][fCol] * N[inRow*width + inCol];
            }
        }
    }

    P[outRow][outCol] = Pvalue;
}




    