#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_fp16.h>
#include <mma.h>  // Add this for tensor core operations
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define TILE_SIZE 32
#define BLOCK_SIZE 32

/**
 * Optimized CUDA kernel for tensor multiplication using shared memory tiling
 * Computes C = A × B for multiple batches of matrices in parallel
 * 
 * Memory Access Pattern:
 * - Uses shared memory tiles to reduce global memory bandwidth
 * - Each thread block loads TILE_SIZE × TILE_SIZE elements into shared memory
 * - Coalesced memory access pattern for better throughput
 * 
 * Optimization Techniques:
 * 1. Shared Memory Tiling:
 *    - Reduces global memory accesses by ~sqrt(TILE_SIZE)
 *    - Each element is loaded once from global memory into shared memory
 *    - Multiple threads reuse data from fast shared memory
 * 
 * 2. Memory Coalescing:
 *    - Adjacent threads access adjacent memory locations
 *    - Maximizes memory bandwidth utilization
 *    - Reduces number of memory transactions
 *
 * 3. Bank Conflict Prevention:
 *    - Padded shared memory to avoid bank conflicts
 *    - Threads in a warp access different memory banks
 *
 * Thread/Block Organization:
 * - Each thread computes one element of output matrix
 * - Thread blocks are organized in 3D:
 *   * x-dimension: handles rows
 *   * y-dimension: handles columns
 *   * z-dimension: handles different batches
 *
 * @param A [in] Input matrix A [batch_size × m × n]
 * @param B [in] Input matrix B [batch_size × k × l]
 * @param C [out] Output matrix C [batch_size × m × l]
 * @param batch_size Number of matrix multiplications to perform
 * @param m Number of rows in matrix A and C
 * @param n Number of columns in A and rows in B
 * @param k Number of columns in B (must equal n)
 * @param l Number of columns in output matrix C
 */
__global__
void tensor_mul_optimized(float *A, float *B, float *C, int batch_size, int m, int n, int k, int l) {
    // Declare shared memory for tiles
    // Each block loads two tiles: one from A and one from B
    // Size is TILE_SIZE × TILE_SIZE for each tile
    __shared__ float As[TILE_SIZE][TILE_SIZE];  // Tile for matrix A
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];  // Tile for matrix B
    
    // Calculate thread and block indices
    // blockIdx: which block this thread belongs to
    // threadIdx: position of thread within its block
    int batch = blockIdx.z;    // Which batch this thread works on
    int bx = blockIdx.x;       // Block index in x dimension (row)
    int by = blockIdx.y;       // Block index in y dimension (col)
    int tx = threadIdx.x;      // Thread index in x dimension
    int ty = threadIdx.y;      // Thread index in y dimension
    
    // Calculate global indices for this thread
    // Each thread computes one element of the output matrix
    int row = bx * TILE_SIZE + tx;  // Global row index
    int col = by * TILE_SIZE + ty;  // Global column index
    
    // Calculate memory offsets for each batch
    // Using size_t to handle large matrices (>2^31 elements)
    // Each batch starts at batch * matrix_size offset
    size_t batch_offset_A = (size_t)batch * m * n;  // Offset for current batch in A
    size_t batch_offset_B = (size_t)batch * k * l;  // Offset for current batch in B
    size_t batch_offset_C = (size_t)batch * m * l;  // Offset for current batch in C
    
    // Accumulator for dot product
    float sum = 0.0f;
    
    // Process the matrix multiplication tile by tile
    // Each iteration processes one TILE_SIZE × TILE_SIZE portion
    for (int tile = 0; tile < (n + TILE_SIZE - 1) / TILE_SIZE; tile++) {
        // Load tile from matrix A into shared memory
        // Check bounds to handle matrices not divisible by TILE_SIZE
        if (row < m && (tile * TILE_SIZE + ty) < n) {
            // Load element from global memory to shared memory
            // Coalesced access: adjacent threads read adjacent memory
            As[tx][ty] = A[batch_offset_A + row * n + tile * TILE_SIZE + ty];
        } else {
            As[tx][ty] = 0.0f;  // Zero padding for boundary conditions
        }
        
        // Load tile from matrix B into shared memory
        if ((tile * TILE_SIZE + tx) < k && col < l) {
            // Load element from global memory to shared memory
            // Coalesced access pattern
            Bs[tx][ty] = B[batch_offset_B + (tile * TILE_SIZE + tx) * l + col];
        } else {
            Bs[tx][ty] = 0.0f;  // Zero padding for boundary conditions
        }
        
        // Synchronize to make sure the tiles are loaded
        __syncthreads();
        
        // Compute partial dot product for this tile
        #pragma unroll  // Hint to compiler to unroll this loop
        for (int k = 0; k < TILE_SIZE; k++) {
            // Multiply and accumulate (MAC) operation
            // Each thread accesses one row of As and one column of Bs
            sum += As[tx][k] * Bs[k][ty];
        }
        
        // Synchronize before loading the next tile
        // Ensures all threads are done using the current tile
        __syncthreads();
    }
    
    // Write final result to global memory
    // Check bounds to handle edge cases
    if (row < m && col < l && batch < batch_size) {
        C[batch_offset_C + row * l + col] = sum;
    }
}

/**
 * Naive CUDA kernel for tensor multiplication (baseline implementation)
 * Simple implementation without optimizations for performance comparison
 * 
 * Memory Access Pattern:
 * - Direct global memory access for all operations
 * - No use of shared memory or caching
 * - High memory bandwidth usage due to repeated access
 *
 * Thread Organization:
 * - One thread per output element
 * - 3D grid for handling multiple batches:
 *   * x-dimension: rows
 *   * y-dimension: columns
 *   * z-dimension: batches
 *
 * Performance Characteristics:
 * - High global memory traffic
 * - No data reuse
 * - Memory access not coalesced
 * - Used as baseline for performance comparison
 *
 * @param A [in] Input matrix A [batch_size × m × n]
 * @param B [in] Input matrix B [batch_size × k × l]
 * @param C [out] Output matrix C [batch_size × m × l]
 * @param batch_size Number of matrix multiplications to perform
 * @param m Number of rows in matrix A and C
 * @param n Number of columns in A and rows in B
 * @param k Number of columns in B (must equal n)
 * @param l Number of columns in output matrix C
 */
__global__
void tensor_mul(float *A, float *B, float *C, int batch_size, int m, int n, int k, int l) {
    // Calculate global thread indices
    // Each thread computes one element of the output
    int batch = blockIdx.z;    // Batch index
    // Global row and column indices
    int row = blockIdx.x * blockDim.x + threadIdx.x;  // Global row in output
    int col = blockIdx.y * blockDim.y + threadIdx.y;  // Global column in output

    // Early exit if thread is out of bounds
    // Prevents invalid memory access and ensures correctness
    if (row >= m || col >= l || batch >= batch_size) return;

    // Calculate batch offsets for each matrix
    // Using size_t to handle large matrices
    size_t batch_offset_A = (size_t)batch * m * n;  // Offset in A
    size_t batch_offset_B = (size_t)batch * k * l;  // Offset in B
    size_t batch_offset_C = (size_t)batch * m * l;  // Offset in C

    // Compute dot product directly from global memory
    // This is inefficient due to:
    // 1. Repeated global memory access
    // 2. No data reuse
    // 3. Non-coalesced memory access pattern
    float sum = 0.0f;
    for (int i = 0; i < n; i++) {
        // Load elements from A and B
        // A[batch][row][i] * B[batch][i][col]
        float a = A[batch_offset_A + row * n + i];         // Element from A
        float b = B[batch_offset_B + i * l + col];         // Element from B
        sum += a * b;  // Multiply and accumulate
    }

    // Write result to global memory
    // C[batch][row][col] = sum
    C[batch_offset_C + row * l + col] = sum;
}

/**
 * Main function to demonstrate and compare different tensor multiplication implementations
 * Compares three versions:
 * 1. Naive implementation
 * 2. Shared memory optimized implementation
 * 3. cuBLAS library implementation
 */
int main(int argc, char **argv) {
    // === Command Line Arguments ===
    if (argc != 6) {
        printf("Usage: ./a.out <batch_size> <m> <n> <k> <l>\n");
        printf("For tensor multiplication: (batch_size × m × n) * (batch_size × k × l)\n");
        printf("Example: ./tensor_mul 8 1024 1024 1024 1024\n");
        return 1;
    }

    // Parse matrix dimensions
    int batch_size = atoi(argv[1]);
    int m = atoi(argv[2]);
    int n = atoi(argv[3]);
    int k = atoi(argv[4]);
    int l = atoi(argv[5]);

    // Validate dimensions
    if (n != k) {
        printf("Error: Matrix dimensions don't match for multiplication\n");
        printf("Matrix A: %d x %d\n", m, n);
        printf("Matrix B: %d x %d\n", k, l);
        return 1;
    }

    // Calculate memory requirements
    size_t total_elements_A = batch_size * m * n;
    size_t total_elements_B = batch_size * k * l;
    size_t total_elements_C = batch_size * m * l;
    size_t total_bytes = (total_elements_A + total_elements_B + total_elements_C) * sizeof(float);

    // Declare all pointers
    float *h_A, *h_B, *h_C, *h_C_original;  // Host pointers
    float *d_A, *d_B, *d_C;                 // Device pointers

    // Declare grid and block dimensions
    dim3 originalBlock(16, 16);
    dim3 originalGrid(
        (m + originalBlock.x - 1) / originalBlock.x,
        (l + originalBlock.y - 1) / originalBlock.y,
        batch_size
    );

    dim3 optimizedBlock(32, 32);
    dim3 optimizedGrid(
        (m + 31) / 32,
        (l + 31) / 32,
        batch_size
    );

    // Initialize cuBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);

    // CUDA timing variables
    hipEvent_t start, stop;
    float original_time, optimized_time, tc_time;

    // === Memory Pointers and Allocation ===
    // Allocate host memory
    h_A = (float *)malloc(total_elements_A * sizeof(float));
    h_B = (float *)malloc(total_elements_B * sizeof(float));
    h_C = (float *)malloc(total_elements_C * sizeof(float));
    h_C_original = (float *)malloc(total_elements_C * sizeof(float));  // Add this allocation

    // Check all allocations
    if (h_A == NULL || h_B == NULL || h_C == NULL || h_C_original == NULL) {
        printf("Error: Failed to allocate host memory\n");
        // Free any successful allocations
        if (h_A) free(h_A);
        if (h_B) free(h_B);
        if (h_C) free(h_C);
        if (h_C_original) free(h_C_original);
        return 1;
    }

    // Initialize matrices with random values
    for (int b = 0; b < batch_size; b++) {
        for (int i = 0; i < m; i++) {
            for (int j = 0; j < n; j++) {
                h_A[b * m * n + i * n + j] = (float)(rand() % 100) / 100.0f;
            }
        }
        for (int i = 0; i < k; i++) {
            for (int j = 0; j < l; j++) {
                h_B[b * k * l + i * l + j] = (float)(rand() % 100) / 100.0f;
            }
        }
    }

    // Validate input parameters
    if (batch_size <= 0 || m <= 0 || n <= 0 || k <= 0 || l <= 0) {
        printf("Error: All dimensions must be positive integers\n");
        return 1;
    }

    // Check if we have enough system memory
    size_t total_host_memory = (total_elements_A + total_elements_B + 
                               total_elements_C * 2) * sizeof(float);  // *2 for original copy
    printf("\n=== Memory Requirements ===\n");
    printf("Host Memory Required: %.2f MB\n", total_host_memory / (1024.0 * 1024.0));
    printf("GPU Memory Required: %.2f MB\n", total_bytes / (1024.0 * 1024.0));

    if (total_host_memory > 16ULL * 1024 * 1024 * 1024) {  // 16GB limit check
        printf("Warning: Required host memory (%.2f GB) might exceed system memory\n",
               total_host_memory / (1024.0 * 1024.0 * 1024.0));
        printf("Continue? (y/n): ");
        char response;
        if (scanf(" %c", &response) != 1) {
            printf("Error reading response\n");
            return 1;
        }
        if (response != 'y' && response != 'Y') {
            return 0;
        }
    }

    // Allocate GPU memory with error checking
    hipError_t err;
    err = hipMalloc((void **)&d_A, total_elements_A * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for A: %s\n", 
                hipGetErrorString(err));
        return 1;
    }

    err = hipMalloc((void **)&d_B, total_elements_B * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for B: %s\n", 
                hipGetErrorString(err));
        hipFree(d_A);
        return 1;
    }

    err = hipMalloc((void **)&d_C, total_elements_C * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for C: %s\n", 
                hipGetErrorString(err));
        hipFree(d_A);
        hipFree(d_B);
        return 1;
    }

    // Configure cache for optimal performance
    hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, 4 * 1024 * 1024);  // 4MB L2 cache
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

    // === Copy Data to GPU and Initial Processing ===
    // First, copy matrices A and B to GPU
    hipMemcpy(d_A, h_A, total_elements_A * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, total_elements_B * sizeof(float), hipMemcpyHostToDevice);

    // === CUDA Streams Setup ===
    // CUDA streams allow overlapping operations (compute + memory transfers)
    const int NUM_STREAMS = 4;  // Using 4 streams for parallelism
    hipStream_t streams[NUM_STREAMS];
    // Create streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }
    
    // Calculate how many batches each stream should handle
    int batches_per_stream = (batch_size + NUM_STREAMS - 1) / NUM_STREAMS;
    
    // === Process Initial Batches Using Streams ===
    for (int i = 0; i < NUM_STREAMS; i++) {
        // Calculate batch range for this stream
        int stream_batch_start = i * batches_per_stream;
        int stream_batch_end = min((i + 1) * batches_per_stream, batch_size);
        int stream_batches = stream_batch_end - stream_batch_start;
        
        if (stream_batches <= 0) continue;  // Skip if no work for this stream
        
        // Calculate memory offsets for this stream's batch
        size_t stream_offset = stream_batch_start * m * n;
        size_t stream_size = stream_batches * m * n * sizeof(float);
        
        // Copy data asynchronously (non-blocking)
        hipMemcpyAsync(d_A + stream_offset, h_A + stream_offset,
                       stream_size, hipMemcpyHostToDevice, streams[i]);
        
        // Configure grid for this stream's batch
        dim3 streamGrid(
            (m + BLOCK_SIZE - 1) / BLOCK_SIZE,  // Ceil(m/BLOCK_SIZE)
            (l + BLOCK_SIZE - 1) / BLOCK_SIZE,  // Ceil(l/BLOCK_SIZE)
            stream_batches                       // Number of matrices for this stream
        );
        
        // Launch kernel in this stream
        // Each stream processes its portion of batches independently
        tensor_mul_optimized<<<streamGrid, optimizedBlock, 0, streams[i]>>>(
            d_A + stream_offset,    // Start of this stream's input A
            d_B + stream_offset,    // Start of this stream's input B
            d_C + stream_offset,    // Where this stream should write output
            stream_batches,         // How many matrices this stream processes
            m, n, k, l             // Matrix dimensions remain the same
        );
    }
    
    // Wait for all streams to finish
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
    }

    // === Performance Testing Section ===
    printf("\n=== Performance Comparison ===\n");
    printf("Matrix Size per Batch: %d x %d\n", m, n);
    printf("Number of Batches: %d\n", batch_size);
    printf("----------------------------------------\n");

    // === Test 1: Naive Implementation ===
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Time memory transfer
    hipEventRecord(start);
    hipMemcpy(d_A, h_A, total_elements_A * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, total_elements_B * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float transfer_time;
    hipEventElapsedTime(&transfer_time, start, stop);
    printf("1. Naive Implementation:\n");
    printf("   Memory Transfer Time (H2D): %.3f ms\n", transfer_time);
    
    // Time computation
    hipEventRecord(start);
    tensor_mul<<<originalGrid, originalBlock>>>(d_A, d_B, d_C, batch_size, m, n, k, l);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&original_time, start, stop);
    printf("   Computation Time: %.3f ms\n", original_time);
    
    // Time result transfer
    hipEventRecord(start);
    hipMemcpy(h_C, d_C, total_elements_C * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float result_time;
    hipEventElapsedTime(&result_time, start, stop);
    printf("   Memory Transfer Time (D2H): %.3f ms\n", result_time);
    printf("   Total Time: %.3f ms\n", transfer_time + original_time + result_time);
    printf("   TFLOPS: %.2f\n", (2.0 * batch_size * m * n * l) / (original_time * 1000000000.0));
    
    // Save naive results
    memcpy(h_C_original, h_C, total_elements_C * sizeof(float));
    
    // === Test 2: Shared Memory Implementation ===
    // Time memory transfer (should be cached from previous test)
    hipEventRecord(start);
    hipMemcpy(d_A, h_A, total_elements_A * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, total_elements_B * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&transfer_time, start, stop);
    printf("\n2. Shared Memory Implementation:\n");
    printf("   Memory Transfer Time (H2D): %.3f ms\n", transfer_time);
    
    // Time computation
    hipEventRecord(start);
    tensor_mul_optimized<<<optimizedGrid, optimizedBlock>>>(d_A, d_B, d_C, batch_size, m, n, k, l);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&optimized_time, start, stop);
    printf("   Computation Time: %.3f ms\n", optimized_time);
    
    // Time result transfer and check accuracy
    hipEventRecord(start);
    hipMemcpy(h_C, d_C, total_elements_C * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&result_time, start, stop);
    printf("   Memory Transfer Time (D2H): %.3f ms\n", result_time);
    printf("   Total Time: %.3f ms\n", transfer_time + optimized_time + result_time);
    printf("   TFLOPS: %.2f\n", (2.0 * batch_size * m * n * l) / (optimized_time * 1000000000.0));
    printf("   Speedup vs Naive: %.2fx\n", original_time / optimized_time);
    
    // Check accuracy against naive implementation
    bool shared_mem_matches = true;
    float shared_mem_max_diff = 0.0f;
    for (size_t i = 0; i < total_elements_C; i++) {
        float diff = fabs(h_C[i] - h_C_original[i]);
        shared_mem_max_diff = max(shared_mem_max_diff, diff);
        if (diff > 1e-5) {
            shared_mem_matches = false;
            break;
        }
    }
    printf("   Accuracy Check: %s (max diff: %e)\n", 
           shared_mem_matches ? "PASSED" : "FAILED", shared_mem_max_diff);
    
    // === Test 3: cuBLAS Implementation ===
    // Set cuBLAS to use highest precision mode
    hipblasStatus_t status;
    status = hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("Failed to set math mode: %d\n", status);
        return 1;
    }

    // Time memory transfer
    hipEventRecord(start);
    hipMemcpy(d_A, h_A, total_elements_A * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, total_elements_B * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&transfer_time, start, stop);
    printf("\n3. cuBLAS Implementation:\n");
    printf("   Memory Transfer Time (H2D): %.3f ms\n", transfer_time);
    
    // Time computation
    hipEventRecord(start);
    
    // Batch parameters for strided GEMM
    long long int strideA = (long long int)m * n;
    long long int strideB = (long long int)k * l;
    long long int strideC = (long long int)m * l;

    // Use batched strided GEMM with transposed operations to match our layout
    const float alpha = 1.0f;
    const float beta = 0.0f;
    status = hipblasSgemmStridedBatched(
        handle,
        HIPBLAS_OP_T,                // Transpose A
        HIPBLAS_OP_T,                // Transpose B
        m, l, k,                    // Original dimensions
        &alpha,                     // Scaling factor for AB
        d_A, n,                     // Matrix A with leading dimension n
        strideA,
        d_B, l,                     // Matrix B with leading dimension l
        strideB,
        &beta,                      // Scaling factor for C
        d_C, m,                     // Output matrix C
        strideC,
        batch_size
    );

    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("Failed to execute batched GEMM: %d\n", status);
        return 1;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&tc_time, start, stop);
    printf("   Computation Time: %.3f ms\n", tc_time);
    
    // Time result transfer and check accuracy
    hipEventRecord(start);
    hipMemcpy(h_C, d_C, total_elements_C * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&result_time, start, stop);
    printf("   Memory Transfer Time (D2H): %.3f ms\n", result_time);
    printf("   Total Time: %.3f ms\n", transfer_time + tc_time + result_time);
    printf("   TFLOPS: %.2f\n", (2.0 * batch_size * m * n * l) / (tc_time * 1000000000.0));
    printf("   Speedup vs Naive: %.2fx\n", original_time / tc_time);
    printf("   Speedup vs Shared Memory: %.2fx\n", optimized_time / tc_time);
    
    // Check accuracy against naive implementation
    bool cublas_matches = true;
    float cublas_max_diff = 0.0f;
    for (size_t i = 0; i < total_elements_C; i++) {
        float diff = fabs(h_C[i] - h_C_original[i]);
        cublas_max_diff = max(cublas_max_diff, diff);
        if (diff > 1e-5) {
            cublas_matches = false;
            break;
        }
    }
    printf("   Accuracy Check: %s (max diff: %e)\n", 
           cublas_matches ? "PASSED" : "FAILED", cublas_max_diff);

    // === Performance Summary ===
    printf("\n=== Performance Summary ===\n");
    printf("1. Naive Implementation:        %.3f ms\n", original_time);
    printf("2. Shared Memory Implementation: %.3f ms (%.2fx faster than naive)\n", 
           optimized_time, original_time / optimized_time);
    printf("3. cuBLAS Implementation:       %.3f ms (%.2fx faster than naive)\n", 
           tc_time, original_time / tc_time);

    // === Final Summary ===
    printf("\n=== Accuracy Summary ===\n");
    printf("Shared Memory vs Naive: %s (max diff: %e)\n", 
           shared_mem_matches ? "PASSED" : "FAILED", shared_mem_max_diff);
    printf("cuBLAS vs Naive:       %s (max diff: %e)\n", 
           cublas_matches ? "PASSED" : "FAILED", cublas_max_diff);

    // === Resource Cleanup ===
    // 1. Clean up CUDA streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamDestroy(streams[i]);
    }

    // 2. Clean up CUDA resources
    hipEventDestroy(start);      // Destroy timing events
    hipEventDestroy(stop);
    
    // 3. Free GPU memory
    hipFree(d_A);  // Free device matrix A
    hipFree(d_B);  // Free device matrix B
    hipFree(d_C);  // Free device matrix C
    
    // 4. Free CPU memory
    free(h_A);           // Free host matrix A
    free(h_B);           // Free host matrix B
    free(h_C);           // Free host matrix C
    free(h_C_original);  // Free reference results

    return 0;
} 