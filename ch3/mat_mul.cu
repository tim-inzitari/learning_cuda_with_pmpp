#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__
void mat_mul(float *A, float *B, float *C, int m, int n, int k, int l){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < m && col < l){
        float sum = 0.0f;
        for (int i = 0; i < n; i++){
            sum += A[row * n + i] * B[i * l + col];
        }
        C[row * l + col] = sum;
    }
    
    
}


int main(int argc, char **argv)
{

    float elapsed_time;
    hipEvent_t start, stop;
    clock_t cpu_start, cpu_end;

    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    if (argc != 5){
        printf("Usage: ./a.out <m> <n> <k> <l>\n");
        return 1;
    }

    int m = atoi(argv[1]);
    int n = atoi(argv[2]);
    int k = atoi(argv[3]);
    int l = atoi(argv[4]);

    printf("Matrix size: %d x %d\n", m, n);
    printf("Matrix size: %d x %d\n", k, l);
    printf("----------------------------------------\n");   

    // Allocate memory on host
    h_A = (float *)malloc(m * n * sizeof(float));
    h_B = (float *)malloc(k * l * sizeof(float));
    h_C = (float *)malloc(m * l * sizeof(float));

    // Initialize host memory   
    for (int i = 0; i < m; i++){
        for (int j = 0; j < n; j++){
            h_A[i * n + j] = (float)(rand() % 100);
        }
    }
    
    // Calculate total GPU memory required
    size_t total_gpu_memory = (m * n + k * l + m * l) * sizeof(float);
    printf("Total GPU memory required: %.2f MB\n", total_gpu_memory / (1024.0 * 1024.0));

    // Allocate memory on device
    hipError_t err;
    err = hipMalloc((void **)&d_A, m * n * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for d_A (%.2f MB) - %s\n", 
                (m * n * sizeof(float)) / (1024.0 * 1024.0),
                hipGetErrorString(err));
        return 1;
    }

    err = hipMalloc((void **)&d_B, k * l * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for d_B (%.2f MB) - %s\n",
                (k * l * sizeof(float)) / (1024.0 * 1024.0),
                hipGetErrorString(err));
        hipFree(d_A);
        return 1;
    }

    err = hipMalloc((void **)&d_C, m * l * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for d_C (%.2f MB) - %s\n",
                (m * l * sizeof(float)) / (1024.0 * 1024.0),
                hipGetErrorString(err));
        hipFree(d_A);
        hipFree(d_B);
        return 1;
    }

    // Copy host memory to device memory
    hipMemcpy(d_A, h_A, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k * l * sizeof(float), hipMemcpyHostToDevice);

    // Create CUDA events for timing
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start timing 
    cpu_start = clock();
    hipEventRecord(start);

    // Launch the kernel
    dim3 dimBlock(32, 32);  
    dim3 dimGrid((m + dimBlock.x - 1) / dimBlock.x, 
                 (l + dimBlock.y - 1) / dimBlock.y);
    mat_mul<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, m, n, k, l);

    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    cpu_end = clock();

    // Copy device memory to host memory
    hipMemcpy(h_C, d_C, m * l * sizeof(float), hipMemcpyDeviceToHost);

    // Print timing information
    printf("----------------------------------------\n");
    printf("Performance Results:\n");
    printf("GPU Kernel Time: %f ms\n", elapsed_time);
    printf("Total CPU Time: %f ms\n", 1000.0 * (double)(cpu_end - cpu_start) / CLOCKS_PER_SEC);

    // Cleanup timing events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}